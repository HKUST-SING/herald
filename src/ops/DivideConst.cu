#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void div_const_kernel(const float *input, float *output, float value,
                                 size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = value / input[ind];
}

int DLGpuMatrixDivConst(float val, const DLArrayHandle input,
                        DLArrayHandle output,
                        DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *output_data = (float *)output->data;
    const float *input_data = (const float *)input->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        div_const_kernel<<<blocks, threads, 0,
                           *(hipStream_t *)stream_handle->handle>>>(
            input_data, output_data, val, size);
    else
        div_const_kernel<<<blocks, threads>>>(input_data, output_data, val,
                                              size);
    return 0;
}