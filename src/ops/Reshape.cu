#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

extern __global__ void float_memory_copy(float *A, const float *B, size_t len);

int DLGpuReshape(const DLArrayHandle in_arr, DLArrayHandle out_arr,
                 DLStreamHandle stream_handle = NULL) {
    size_t input_size = 1;
    size_t output_size = 1;
    for (index_t i = 0; i < in_arr->ndim; i++) {
        input_size *= in_arr->shape[i];
    }
    for (index_t i = 0; i < out_arr->ndim; i++) {
        output_size *= out_arr->shape[i];
    }
    assert(input_size == output_size);
    const float *input_data = (const float *)in_arr->data;
    float *output_data = (float *)out_arr->data;
    size_t BLOCKS = (input_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    if (stream_handle)
        float_memory_copy<<<BLOCKS, THREADS_PER_BLOCK, 0,
                            *(hipStream_t *)stream_handle->handle>>>(
            output_data, input_data, input_size);
    else
        float_memory_copy<<<BLOCKS, THREADS_PER_BLOCK>>>(
            output_data, input_data, input_size);
    return 0;
}