#include "gpu_runtime.h"

int DLGpuBatchMatrixMultiply(const DLArrayHandle matA, bool transposeA,
                             const DLArrayHandle matB, bool transposeB,
                             DLArrayHandle matC,
                             DLStreamHandle stream_handle = NULL) {
    assert(matA->ndim == matB->ndim);
    assert(matA->ndim == matC->ndim);

    int dev_id = (matA->ctx).device_id;
    cublas_init(dev_id, stream_handle);

    float one = 1.0f;
    float zero = 0.0f;

    int ndim = matA->ndim;
    int m = matC->shape[ndim - 1];
    int n = matC->shape[ndim - 2];
    int k = transposeA ? matA->shape[ndim - 2] : matA->shape[ndim - 1];
    long long int strideA = matA->shape[ndim - 2] * matA->shape[ndim - 1];
    long long int strideB = matB->shape[ndim - 2] * matB->shape[ndim - 1];
    long long int strideC = matC->shape[ndim - 2] * matC->shape[ndim - 1];

    int batchCount = 1;
    for (int i = 0; i < ndim - 2; ++i) {
        assert(matA->shape[i] == matB->shape[i]);
        assert(matA->shape[i] == matC->shape[i]);
        batchCount *= matA->shape[i];
    }

    hipblasStatus_t res = hipblasSgemmStridedBatched(
        cublas_map[dev_id], transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
        transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N, m, n, k, &one,
        (const float *)matB->data, !transposeB ? m : k, strideB,
        (const float *)matA->data, !transposeA ? k : n, strideA, &zero,
        (float *)matC->data, m, strideC, batchCount);
    assert(res == HIPBLAS_STATUS_SUCCESS);
    return 0;
}
