#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void cudnn_cross_entropy_kernel(const float *logsoftmax,
                                           const float *label, float *output,
                                           size_t size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
        return;
    output[idx] = -logsoftmax[idx] * label[idx];
}

int CuDNN_DLGpuSoftmaxEntropy(const DLArrayHandle input_y,
                              const DLArrayHandle label, DLArrayHandle output,
                              DLStreamHandle stream_handle = NULL) {
    size_t indim = input_y->ndim;
    assert(indim == label->ndim && indim == output->ndim + 1);
    int dev_id = (input_y->ctx).device_id;
    cudnn_init(dev_id, stream_handle);
    int n_ = 1;
    for (int i = 0; i < indim - 1; ++i) {
        n_ *= input_y->shape[i];
    }
    int c_ = input_y->shape[indim - 1];
    size_t size = n_ * c_;
    const float *y_data = (const float *)(input_y->data);
    float *label_data = (float *)(label->data);
    float *output_data = (float *)(output->data);
    float alpha = 1.0f;
    float beta = 0.0f;
    hipdnnTensorDescriptor_t desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, n_, c_, 1, 1));
    if (is_chunk_init(dev_id) == false) {
        chunk_init(dev_id);
    }
    void *temp_data = find_chunk(size * sizeof(float), dev_id);
    CUDNN_CALL(hipdnnSoftmaxForward(
        cudnn_map[dev_id], HIPDNN_SOFTMAX_LOG, HIPDNN_SOFTMAX_MODE_INSTANCE,
        &alpha, desc, (const void *)y_data, &beta, desc, temp_data));

    dim3 blocks;
    dim3 threads;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle) {
        cudnn_cross_entropy_kernel<<<blocks, threads, 0,
                                     *(hipStream_t *)stream_handle->handle>>>(
            (const float *)temp_data, label_data, (float *)temp_data, size);
    } else {
        cudnn_cross_entropy_kernel<<<blocks, threads>>>(
            (const float *)temp_data, label_data, (float *)temp_data, size);
    }

    hipdnnReduceTensorDescriptor_t rtd;
    CUDNN_CALL(hipdnnCreateReduceTensorDescriptor(&rtd));
    CUDNN_CALL(hipdnnSetReduceTensorDescriptor(
        rtd, HIPDNN_REDUCE_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN,
        HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES));
    hipdnnTensorDescriptor_t new_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&new_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(new_desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, n_, 1, 1, 1));
    CUDNN_CALL(hipdnnReduceTensor(cudnn_map[dev_id], rtd, NULL, 0, temp_data,
                                 size * sizeof(float), &alpha, desc,
                                 (const void *)temp_data, &beta, new_desc,
                                 (void *)output_data));

    CUDNN_CALL(hipdnnDestroyReduceTensorDescriptor(rtd));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(new_desc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
    del_chunk(temp_data, dev_id);
    return 0;
}

__global__ void
cudnn_softmax_cross_entropy_gradient(const float *pred, const float *y_,
                                     const float *grad_data, float *output_data,
                                     int last_dim, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output_data[ind] = (pred[ind] - y_[ind]) * grad_data[ind / last_dim];
}

int CuDNN_DLGpuSoftmaxEntropyGradient(const DLArrayHandle grad,
                                      const DLArrayHandle input_y,
                                      const DLArrayHandle label,
                                      DLArrayHandle output,
                                      DLStreamHandle stream_handle = NULL) {
    size_t indim = input_y->ndim;
    assert(indim == label->ndim && indim == output->ndim
           && indim == grad->ndim + 1);
    int dev_id = (input_y->ctx).device_id;
    cudnn_init(dev_id, stream_handle);
    int n_ = 1;
    for (int i = 0; i < indim - 1; ++i) {
        n_ *= input_y->shape[i];
    }
    int c_ = input_y->shape[indim - 1];
    size_t size = n_ * c_;
    const float *grad_data = (const float *)grad->data;
    const float *y_data = (const float *)input_y->data;
    const float *label_data = (const float *)label->data;
    float *output_data = (float *)output->data;
    if (is_chunk_init(dev_id) == false) {
        chunk_init(dev_id);
    }
    void *temp_data = find_chunk(size * sizeof(float), dev_id);
    float alpha = 1.0f;
    float beta = 0.0f;
    hipdnnTensorDescriptor_t desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, n_, c_, 1, 1));
    CUDNN_CALL(hipdnnSoftmaxForward(
        cudnn_map[dev_id], HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE,
        &alpha, desc, (const void *)y_data, &beta, desc, temp_data));
    dim3 blocks;
    dim3 threads;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle) {
        cudnn_softmax_cross_entropy_gradient<<<
            blocks, threads, 0, *(hipStream_t *)stream_handle->handle>>>(
            (const float *)temp_data, label_data, grad_data, output_data, c_,
            size);
    } else {
        cudnn_softmax_cross_entropy_gradient<<<blocks, threads>>>(
            (const float *)temp_data, label_data, grad_data, output_data, c_,
            size);
    }
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
    del_chunk(temp_data, dev_id);
    return 0;
}
