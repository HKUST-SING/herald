#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void sqrt_kernel(float *input, float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = sqrtf(input[ind]);
}

int DLGpuSqrt(const DLArrayHandle input, DLArrayHandle output,
              DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *input_data = (float *)input->data;
    float *output_data = (float *)output->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        sqrt_kernel<<<blocks, threads, 0,
                      *(hipStream_t *)stream_handle->handle>>>(
            input_data, output_data, size);
    else
        sqrt_kernel<<<blocks, threads>>>(input_data, output_data, size);
    return 0;
}

__global__ void reciprocal_sqrt_kernel(float *input, float *output,
                                       size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = rsqrtf(input[ind]);
}

int DLGpuReciprocalSqrt(const DLArrayHandle input, DLArrayHandle output,
                        DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *input_data = (float *)input->data;
    float *output_data = (float *)output->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        reciprocal_sqrt_kernel<<<blocks, threads, 0,
                                 *(hipStream_t *)stream_handle->handle>>>(
            input_data, output_data, size);
    else
        reciprocal_sqrt_kernel<<<blocks, threads>>>(input_data, output_data,
                                                    size);
    return 0;
}