#include "gpu_runtime.h"
#include <nccl.h>

int CuSparse_DLGpuCsrmv(const DLArrayHandle data_handle,
                        const DLArrayHandle row_handle,
                        const DLArrayHandle col_handle, int nrow, int ncol,
                        bool transpose, const DLArrayHandle input_handle,
                        DLArrayHandle output_handle,
                        DLStreamHandle stream_handle = NULL) {
    assert(data_handle->ndim == 1);
    assert(row_handle->ndim == 1);
    assert(col_handle->ndim == 1);
    assert(transpose ? nrow == input_handle->shape[0] :
                       ncol == input_handle->shape[0]);

    int nnz = data_handle->shape[0];
    int dev_id = (data_handle->ctx).device_id;
    cusp_init(dev_id, stream_handle);

    float alpha = 1.0;
    float beta = 0.0;

    hipsparseMatDescr_t descr = 0;
    CUSP_CALL(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseOperation_t trans = transpose ? HIPSPARSE_OPERATION_TRANSPOSE :
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE;
    // CUSP_CALL(hipsparseScsrmv(
    //     cusp_map[dev_id], trans, nrow, ncol, nnz, (const float *)&alpha,
    //     descr, (const float *)data_handle->data, (const int
    //     *)row_handle->data, (const int *)col_handle->data, (const float
    //     *)input_handle->data, (const float *)&beta, (float
    //     *)output_handle->data));
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void *dBuffer = nullptr;
    size_t buffer_size = 0;
    CUSP_CALL(hipsparseCreateCsr(&matA, nrow, ncol, nnz, row_handle->data,
                      col_handle->data, data_handle->data, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    CUSP_CALL(hipsparseCreateDnVec(&vecX, ncol, input_handle->data, HIP_R_32F));
    CUSP_CALL(hipsparseCreateDnVec(&vecY, nrow, output_handle->data, HIP_R_32F));

    hipsparseSpMV_bufferSize(cusp_map[dev_id], trans, (const float*)&alpha, matA, vecX, (const float *)&beta, vecY, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, &buffer_size);
    CUSP_CALL(hipsparseSpMV(cusp_map[dev_id], HIPSPARSE_OPERATION_NON_TRANSPOSE,
                           &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                           HIPSPARSE_MV_ALG_DEFAULT, dBuffer));

    CUSP_CALL(hipsparseDestroySpMat(matA));
    CUSP_CALL(hipsparseDestroyDnVec(vecX));
    CUSP_CALL(hipsparseDestroyDnVec(vecY));

    return 0;
}