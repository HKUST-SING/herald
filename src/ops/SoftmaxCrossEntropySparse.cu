#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)
__global__ void matrix_softmax_cross_entropy_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    const float *input_b,
                                                    const int ignored_index,                                                 
                                                    float *output) {
    // Two dimensional thread blocks.
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= nrow) return;
    if(int(input_b[id])==ignored_index)
    {
        output[id] = 0;
        return;
    }
            
    float maxval = input_a[id * ncol];
    // Find max for a row.
    for (int x = 1; x < ncol; ++x) {
        maxval = max(maxval, input_a[id * ncol + x]);
    }
    // Deduct by max for a row, and raise to exp.
    float sum = 0;
    for (int x = 0; x < ncol; ++x) {
        sum += exp(input_a[id * ncol + x] - maxval);
    }
    // Compute per-row loss.
    size_t curid = id * ncol + int(input_b[id]);
    float loss = -(input_a[curid] - maxval) + log(sum);
    output[id] = loss;
}

int DLGpuSoftmaxCrossEntropySparse(const DLArrayHandle input_a,
                             const DLArrayHandle input_b,
                             const int ignored_index,
                             DLArrayHandle output, DLStreamHandle stream_handle = NULL) {
    size_t indim = input_a->ndim;
    
    assert (output->ndim == input_b->ndim && indim == output->ndim + 1);
    int nrow = 1;
    for (int i = 0; i < indim-1; ++i) {
        assert (input_a->shape[i] == input_b->shape[i] &&
            input_a->shape[i] == output->shape[i]);
        nrow *= input_a->shape[i];
    }

    int ncol = input_a->shape[indim-1];

    const float *input_data_a = (const float *)input_a->data;
    const float *input_data_b = (const float *)input_b->data;
    float *output_data = (float *)output->data;

    dim3 blocks;
    dim3 threads;
    if (nrow <= 1024) {
        threads.x = nrow;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (nrow + 1023) / 1024;
    }
    // 1 block
    if (stream_handle) {
        matrix_softmax_cross_entropy_kernel<<<blocks, threads, 0, *(hipStream_t*)stream_handle->handle>>>(
            nrow, ncol, input_data_a, input_data_b, ignored_index, output_data);
    } else {
        matrix_softmax_cross_entropy_kernel<<<blocks, threads>>>(
            nrow, ncol, input_data_a, input_data_b, ignored_index, output_data);
    }
    return 0;
}

__global__ void softmax_cross_entropy_gradient_kernel(int nrow, int ncol, const float *input_a, const float *input_b, const float *input_c, const int ignored_index, float *output) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= nrow) return;

    if(int(input_b[id])==ignored_index)
    {
        for (int x = 0; x < ncol; ++x) 
        {
            size_t curid = id * ncol + x;
            output[curid] = 0;
        }
        return;        
    }
    
    float maxval = input_a[id * ncol];
    // Find max for a row.
    for (int x = 1; x < ncol; ++x) {
        maxval = max(maxval, input_a[id * ncol + x]);
    }
    // Deduct by max for a row, and raise to exp.
    float sum = 0;
    for (int x = 0; x < ncol; ++x) {
        sum += exp(input_a[id * ncol + x] - maxval);
    }
    for (int x = 0; x < ncol; ++x) {
        size_t curid = id * ncol + x;
        if(x == int(input_b[id]))
        {
            output[curid] = (exp(input_a[curid] - maxval) / sum - 1.0) * input_c[id];
        }
        else
            output[curid] = (exp(input_a[curid] - maxval) / sum) * input_c[id];
    }
}

int DLGpuSoftmaxCrossEntropySparse_Gradient(const DLArrayHandle input_a, const DLArrayHandle input_b,
                                      const DLArrayHandle input_c, const int ignored_index,
                                      DLArrayHandle output,
                                      DLStreamHandle stream_handle = NULL) {
    size_t indim = input_a->ndim;
    assert (indim >= 2 && input_c->ndim == input_b->ndim && indim == input_c->ndim + 1 && indim == output->ndim);
    int nrow = 1;
    for (int i = 0; i < indim-1; ++i) {
        assert (input_a->shape[i] == input_b->shape[i] &&
            input_a->shape[i] == output->shape[i] &&
            input_a->shape[i] == input_c->shape[i]);
        nrow *= input_a->shape[i];
    }
    assert (input_a->shape[indim-1] == output->shape[indim-1]);
    int ncol = input_a->shape[indim-1];
        
    const float *input_data_a = (const float *)input_a->data;
    const float *input_data_b = (const float *)input_b->data;
    const float *input_data_c = (const float *)input_c ->data;   
    float *output_data = (float *)output->data;

    dim3 blocks;
    dim3 threads;
    if (nrow <= 1024) {
        threads.x = nrow;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (nrow + 1023) / 1024;
    }
    if (stream_handle) {
        softmax_cross_entropy_gradient_kernel<<<blocks, threads, 0, *(hipStream_t*)stream_handle->handle>>>(
            nrow, ncol, input_data_a, input_data_b, input_data_c, ignored_index, output_data);
    } else {
        softmax_cross_entropy_gradient_kernel<<<blocks, threads>>>(
            nrow, ncol, input_data_a, input_data_b, input_data_c, ignored_index, output_data);
    }
    return 0;
}
