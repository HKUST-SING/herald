#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void ele_mult_kernel(const float *matA, const float *matB,
                                float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = matA[ind] * matB[ind];
}

int DLGpuMatrixElementwiseMultiply(const DLArrayHandle matA,
                                   const DLArrayHandle matB,
                                   DLArrayHandle output,
                                   DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < matA->ndim; i++) {
        size *= matA->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *output_data = (float *)output->data;
    const float *matA_data = (const float *)matA->data;
    const float *matB_data = (const float *)matB->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        ele_mult_kernel<<<blocks, threads, 0,
                          *(hipStream_t *)stream_handle->handle>>>(
            matA_data, matB_data, output_data, size);
    else
        ele_mult_kernel<<<blocks, threads>>>(matA_data, matB_data, output_data,
                                             size);
    return 0;
}