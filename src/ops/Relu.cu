#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void relu_kernel(float *input, float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = max(input[ind], 0.);
}

int DLGpuRelu(const DLArrayHandle input, DLArrayHandle output,
              DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *input_data = (float *)input->data;
    float *output_data = (float *)output->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        relu_kernel<<<blocks, threads, 0,
                      *(hipStream_t *)stream_handle->handle>>>(
            input_data, output_data, size);
    else
        relu_kernel<<<blocks, threads>>>(input_data, output_data, size);
    return 0;
}

__global__ void relu_grad_kernel(const float *input, const float *in_grad,
                                 float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    float s = 0;
    if (input[ind] > 0)
        s = 1;
    if (input[ind] < 0)
        s = -1;
    output[ind] = (s + 1) * in_grad[ind] * 0.5;
}

int DLGpuReluGradient(const DLArrayHandle input, const DLArrayHandle in_grad,
                      DLArrayHandle output,
                      DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    const float *input_data = (const float *)input->data;
    const float *in_grad_data = (const float *)in_grad->data;
    float *output_data = (float *)output->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        relu_grad_kernel<<<blocks, threads, 0,
                           *(hipStream_t *)stream_handle->handle>>>(
            input_data, in_grad_data, output_data, size);
    else
        relu_grad_kernel<<<blocks, threads>>>(input_data, in_grad_data,
                                              output_data, size);
    return 0;
}
