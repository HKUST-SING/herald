#include "gpu_runtime.h"
#include <cudnn_cnn_infer.h>
#include <cudnn_cnn_train.h>

int CuDNN_DLGpuConv2d(const DLArrayHandle input_x, const DLArrayHandle input_f,
                      DLArrayHandle output, const int padding_h,
                      const int padding_w, const int stride_h,
                      const int stride_w, DLStreamHandle stream_handle = NULL) {
    int dev_id = (input_x->ctx).device_id;
    cudnn_init(dev_id, stream_handle);
    size_t input_N = input_x->shape[0];
    size_t input_C = input_x->shape[1];
    size_t input_H = input_x->shape[2];
    size_t input_W = input_x->shape[3];
    const float *input_data = (const float *)input_x->data;

    // input
    hipdnnTensorDescriptor_t input_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, input_N, input_C,
                                          input_H, input_W));
    size_t filter_N = input_f->shape[0];
    size_t filter_C = input_f->shape[1];
    size_t filter_H = input_f->shape[2];
    size_t filter_W = input_f->shape[3];
    const float *filter_data = (const float *)input_f->data;

    // filter
    hipdnnFilterDescriptor_t filter_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT,
                                          HIPDNN_TENSOR_NCHW, filter_N, filter_C,
                                          filter_H, filter_W));

    // convolution
    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
        conv_desc, padding_h, padding_w, stride_h, stride_w, 1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    size_t out_N = output->shape[0];
    size_t out_C = output->shape[1];
    size_t out_H = output->shape[2];
    size_t out_W = output->shape[3];
    // output
    hipdnnTensorDescriptor_t out_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, out_N, out_C, out_H,
                                          out_W));
    float *output_data = (float *)output->data;
    // algorithm
    hipdnnConvolutionFwdAlgo_t algo;
    int fw_alg_cnt = 0;
    CUDNN_CALL(cudnnGetConvolutionForwardAlgorithmMaxCount(cudnn_map[dev_id],
                                                           &fw_alg_cnt));
    hipdnnConvolutionFwdAlgoPerf_t* perf = new hipdnnConvolutionFwdAlgoPerf_t[fw_alg_cnt];
    // algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    // algo = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED;
    // tune this function to work in cudnn8
    int rtn_alg_cnt = 0;
    CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithm(
        cudnn_map[dev_id], input_desc, filter_desc, conv_desc, out_desc,
        fw_alg_cnt, &rtn_alg_cnt, perf));
    // use the fastest algorithm
    algo = perf[0].algo;
    size_t workspace_size;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn_map[dev_id], input_desc, filter_desc, conv_desc, out_desc, algo,
        &workspace_size));

    if (is_chunk_init(dev_id) == false) {
        chunk_init(dev_id);
    }
    void *work_data = find_chunk(workspace_size, dev_id);

    float alpha = 1.0f;
    float beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionForward(
        cudnn_map[dev_id], &alpha, input_desc, input_data, filter_desc,
        filter_data, conv_desc, algo, work_data, workspace_size, &beta,
        out_desc, output_data));
    del_chunk(work_data, dev_id);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    delete[] perf;
    return 0;
}
int CuDNN_DLGpuConv2d_Gradient_of_Filter(const DLArrayHandle input_x,
                                         const DLArrayHandle gradient_y,
                                         DLArrayHandle gradient_f,
                                         const int padding_h,
                                         const int padding_w,
                                         const int stride_h, const int stride_w,
                                         DLStreamHandle stream_handle = NULL) {
    // create handle
    int dev_id = (input_x->ctx).device_id;
    cudnn_init(dev_id, stream_handle);

    // input
    size_t input_N = input_x->shape[0];
    size_t input_C = input_x->shape[1];
    size_t input_H = input_x->shape[2];
    size_t input_W = input_x->shape[3];
    const float *input_data = (const float *)input_x->data;

    hipdnnTensorDescriptor_t input_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, input_N, input_C,
                                          input_H, input_W));
    // dy
    size_t dy_N = gradient_y->shape[0];
    size_t dy_C = gradient_y->shape[1];
    size_t dy_H = gradient_y->shape[2];
    size_t dy_W = gradient_y->shape[3];
    const float *dy_data = (const float *)gradient_y->data;

    hipdnnTensorDescriptor_t dy_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&dy_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
        dy_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dy_N, dy_C, dy_H, dy_W));

    // conv2d
    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
        conv_desc, padding_h, padding_w, stride_h, stride_w, 1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    // dw
    size_t df_N = gradient_f->shape[0];
    size_t df_C = gradient_f->shape[1];
    size_t df_H = gradient_f->shape[2];
    size_t df_W = gradient_f->shape[3];
    float *df_data = (float *)gradient_f->data;

    hipdnnFilterDescriptor_t df_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&df_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(
        df_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, df_N, df_C, df_H, df_W));

    // algo
    hipdnnConvolutionBwdFilterAlgo_t algo;
    int filter_alg_cnt = 0;
    CUDNN_CALL(cudnnGetConvolutionBackwardFilterAlgorithmMaxCount(cudnn_map[dev_id],
                                                           &filter_alg_cnt));
    hipdnnConvolutionBwdFilterAlgoPerf_t *perf =
        new hipdnnConvolutionBwdFilterAlgoPerf_t[filter_alg_cnt];
    int rtn_alg_cnt = 0;
    CUDNN_CALL(hipdnnFindConvolutionBackwardFilterAlgorithm(
        cudnn_map[dev_id], input_desc, dy_desc, conv_desc, df_desc,
        filter_alg_cnt, &rtn_alg_cnt, perf));
    // algo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT;
    // algo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0;
    algo = perf[0].algo;
    size_t workspace_size;
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
        cudnn_map[dev_id], input_desc, dy_desc, conv_desc, df_desc, algo,
        &workspace_size));
    if (is_chunk_init(dev_id) == false) {
        chunk_init(dev_id);
    }
    void *work_data = find_chunk(workspace_size, dev_id);
    float alpha = 1.0f;
    float beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionBackwardFilter(
        cudnn_map[dev_id], &alpha, input_desc, input_data, dy_desc, dy_data,
        conv_desc, algo, work_data, workspace_size, &beta, df_desc, df_data));
    del_chunk(work_data, dev_id);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(dy_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(df_desc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    delete[] perf;
    return 0;
}

int CuDNN_DLGpuConv2d_Gradient_of_Data(const DLArrayHandle input_f,
                                       const DLArrayHandle gradient_y,
                                       DLArrayHandle gradient_x,
                                       const int padding_h, const int padding_w,
                                       const int stride_h, const int stride_w,
                                       DLStreamHandle stream_handle = NULL) {
    // create handle
    int dev_id = (input_f->ctx).device_id;
    cudnn_init(dev_id, stream_handle);

    // filter
    size_t filter_N = input_f->shape[0];
    size_t filter_C = input_f->shape[1];
    size_t filter_H = input_f->shape[2];
    size_t filter_W = input_f->shape[3];
    const float *filter_data = (const float *)input_f->data;

    hipdnnFilterDescriptor_t filter_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT,
                                          HIPDNN_TENSOR_NCHW, filter_N, filter_C,
                                          filter_H, filter_W));
    // dy
    size_t dy_N = gradient_y->shape[0];
    size_t dy_C = gradient_y->shape[1];
    size_t dy_H = gradient_y->shape[2];
    size_t dy_W = gradient_y->shape[3];
    const float *dy_data = (const float *)gradient_y->data;

    hipdnnTensorDescriptor_t dy_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&dy_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
        dy_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dy_N, dy_C, dy_H, dy_W));

    // conv2d
    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
        conv_desc, padding_h, padding_w, stride_h, stride_w, 1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    // dx
    size_t dx_N = gradient_x->shape[0];
    size_t dx_C = gradient_x->shape[1];
    size_t dx_H = gradient_x->shape[2];
    size_t dx_W = gradient_x->shape[3];
    float *dx_data = (float *)gradient_x->data;

    hipdnnTensorDescriptor_t dx_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&dx_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
        dx_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, dx_N, dx_C, dx_H, dx_W));

    // algo
    hipdnnConvolutionBwdDataAlgo_t algo;
    int bw_alg_cnt = 0;
    CUDNN_CALL(cudnnGetConvolutionBackwardDataAlgorithmMaxCount(
        cudnn_map[dev_id], &bw_alg_cnt));
    hipdnnConvolutionBwdDataAlgoPerf_t* perf = new hipdnnConvolutionBwdDataAlgoPerf_t[bw_alg_cnt];
    int rtn_alg_cnt = 0;
    CUDNN_CALL(hipdnnFindConvolutionBackwardDataAlgorithm(
        cudnn_map[dev_id], filter_desc, dy_desc, conv_desc, dx_desc,
        bw_alg_cnt, &rtn_alg_cnt, perf));
    // algo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0;
    // algo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_WINOGRAD_NONFUSED;
    algo = perf[0].algo;
    size_t workspace_size;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(
        cudnn_map[dev_id], filter_desc, dy_desc, conv_desc, dx_desc, algo,
        &workspace_size));
    if (is_chunk_init(dev_id) == false) {
        chunk_init(dev_id);
    }
    void *work_data = find_chunk(workspace_size, dev_id);

    float alpha = 1.0f;
    float beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionBackwardData(
        cudnn_map[dev_id], &alpha, filter_desc, filter_data, dy_desc, dy_data,
        conv_desc, algo, work_data, workspace_size, &beta, dx_desc, dx_data));
    del_chunk(work_data, dev_id);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(dy_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(dx_desc));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
    delete[] perf;
    return 0;
}
