#include "hip/hip_runtime.h"
#include "gpu_runtime.h"
#include <hiprand/hiprand_kernel.h>

__global__ void init_normal_kernel(float *arr, const float mean,
                                   const float stddev, unsigned long long seed,
                                   size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed, 0, ind, &state);
    arr[ind] = hiprand_normal(&state) * stddev + mean;
}

int DLGpuNormalInit(DLArrayHandle arr, const float mean, const float stddev,
                    unsigned long long seed,
                    DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < arr->ndim; i++) {
        size *= arr->shape[i];
    }
    float *arr_data = (float *)arr->data;

    dim3 blocks;
    dim3 threads;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle) {
        init_normal_kernel<<<blocks, threads, 0,
                             *(hipStream_t *)stream_handle->handle>>>(
            arr_data, mean, stddev, seed, size);
    } else {
        init_normal_kernel<<<blocks, threads>>>(arr_data, mean, stddev, seed,
                                                size);
    }

    return 0;
}

__global__ void init_scale_kernel(float *arr, const float lb, const float ub,
                                  unsigned long long seed, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed, 0, ind, &state);
    arr[ind] = hiprand_uniform(&state) * (ub - lb) + lb;
}

int DLGpuUniformInit(DLArrayHandle arr, const float lb, const float ub,
                     unsigned long long seed,
                     DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < arr->ndim; i++) {
        size *= arr->shape[i];
    }
    float *arr_data = (float *)arr->data;

    dim3 blocks;
    dim3 threads;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle) {
        init_scale_kernel<<<blocks, threads, 0,
                            *(hipStream_t *)stream_handle->handle>>>(
            arr_data, lb, ub, seed, size);
    } else {
        init_scale_kernel<<<blocks, threads>>>(arr_data, lb, ub, seed, size);
    }

    return 0;
}

__global__ void truncated_normal_kernel(float *arr, const float mean,
                                        const float stddev,
                                        unsigned long long seed, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    hiprandStatePhilox4_32_10_t state;
    bool not_found = true;
    // here we use different sequences instead of offsets
    // to avoid using the same random number
    hiprand_init(seed, ind, 0, &state);
    float temp;
    while (not_found) {
        temp = hiprand_normal(&state);
        not_found = (temp < -2 || temp > 2);
    }
    arr[ind] = temp * stddev + mean;
}

int DLGpuTruncatedNormalInit(DLArrayHandle arr, const float mean,
                             const float stddev, unsigned long long seed,
                             DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < arr->ndim; i++) {
        size *= arr->shape[i];
    }
    float *arr_data = (float *)arr->data;

    dim3 blocks;
    dim3 threads;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle) {
        truncated_normal_kernel<<<blocks, threads, 0,
                                  *(hipStream_t *)stream_handle->handle>>>(
            arr_data, mean, stddev, seed, size);
    } else {
        truncated_normal_kernel<<<blocks, threads>>>(arr_data, mean, stddev,
                                                     seed, size);
    }

    return 0;
}
