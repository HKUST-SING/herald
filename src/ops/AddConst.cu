#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void add_const_kernel(const float *input, float *output, float value,
                                 size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = input[ind] + value;
}

int DLGpuMatrixElementwiseAddByConst(const DLArrayHandle input, float val,
                                     DLArrayHandle output,
                                     DLStreamHandle stream_handle = NULL) {
    size_t size = 1;
    for (index_t i = 0; i < input->ndim; i++) {
        size *= input->shape[i];
    }
    dim3 blocks;
    dim3 threads;
    float *output_data = (float *)output->data;
    const float *input_data = (const float *)input->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        add_const_kernel<<<blocks, threads, 0,
                           *(hipStream_t *)stream_handle->handle>>>(
            input_data, output_data, val, size);
    else
        add_const_kernel<<<blocks, threads>>>(input_data, output_data, val,
                                              size);
    return 0;
}