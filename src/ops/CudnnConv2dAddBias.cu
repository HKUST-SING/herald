#include "hip/hip_runtime.h"
#include "gpu_runtime.h"


__global__ void conv2d_add_bias(size_t nthreads,
    const float *input_data,
    float *output_data,
    size_t input_size,
    size_t output_size) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= nthreads)
    return;
    size_t input_id = id % input_size / output_size;
    output_data[id] += input_data[input_id];
}


int Cudnn_Conv2dAddBias(const DLArrayHandle input_x, const DLArrayHandle input_f,
                      const DLArrayHandle bias, DLArrayHandle output,
                      const int padding_h, const int padding_w,
                      const int stride_h, const int stride_w,
                      DLStreamHandle stream_handle = NULL) {
    int dev_id = (input_x->ctx).device_id;
    cudnn_init(dev_id, stream_handle);
    size_t input_N = input_x->shape[0];
    size_t input_C = input_x->shape[1];
    size_t input_H = input_x->shape[2];
    size_t input_W = input_x->shape[3];
    const float *input_data = (const float *)input_x->data;

    // input
    hipdnnTensorDescriptor_t input_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, input_N, input_C,
                                          input_H, input_W));
    size_t filter_N = input_f->shape[0];
    size_t filter_C = input_f->shape[1];
    size_t filter_H = input_f->shape[2];
    size_t filter_W = input_f->shape[3];
    const float *filter_data = (const float *)input_f->data;

    // filter
    hipdnnFilterDescriptor_t filter_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT,
                                          HIPDNN_TENSOR_NCHW, filter_N, filter_C,
                                          filter_H, filter_W));

    // convolution
    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
        conv_desc, padding_h, padding_w, stride_h, stride_w, 1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    size_t out_N = output->shape[0];
    size_t out_C = output->shape[1];
    size_t out_H = output->shape[2];
    size_t out_W = output->shape[3];
    // output
    hipdnnTensorDescriptor_t out_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, out_N, out_C, out_H,
                                          out_W));
    float *output_data = (float *)output->data;
    // algorithm
    hipdnnConvolutionFwdAlgo_t algo;
    int fw_alg_cnt = 0;
    CUDNN_CALL(cudnnGetConvolutionForwardAlgorithmMaxCount(cudnn_map[dev_id],
                                                           &fw_alg_cnt));
    hipdnnConvolutionFwdAlgoPerf_t* perf = new hipdnnConvolutionFwdAlgoPerf_t[fw_alg_cnt];
    // algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    // algo = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED;
    // tune this function to work in cudnn8
    int rtn_alg_cnt = 0;
    CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithm(
        cudnn_map[dev_id], input_desc, filter_desc, conv_desc, out_desc,
        fw_alg_cnt, &rtn_alg_cnt, perf));
    // use the fastest algorithm
    algo = perf[0].algo;
    size_t workspace_size;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn_map[dev_id], input_desc, filter_desc, conv_desc, out_desc, algo,
        &workspace_size));

    if (is_chunk_init(dev_id) == false) {
        chunk_init(dev_id);
    }
    void *work_data = find_chunk(workspace_size, dev_id);

    float alpha = 1.0f;
    float beta = 0.0f;
    CUDNN_CALL(hipdnnConvolutionForward(
        cudnn_map[dev_id], &alpha, input_desc, input_data, filter_desc,
        filter_data, conv_desc, algo, work_data, workspace_size, &beta,
        out_desc, output_data));
    del_chunk(work_data, dev_id);
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    
    // add bias
    const float *bias_data = (const float*)bias->data;
    size_t nthreads = out_N * out_C * out_H * out_W;
    size_t BLOCKS = (nthreads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    size_t bias_output_size = out_H * out_W;
    size_t bias_input_size = out_C * bias_output_size;
    if (stream_handle)
        conv2d_add_bias<<<BLOCKS, THREADS_PER_BLOCK, 0,
                                     *(hipStream_t *)stream_handle->handle>>>(
            nthreads, bias_data, output_data, bias_input_size, bias_output_size);
    else
        conv2d_add_bias<<<BLOCKS, THREADS_PER_BLOCK>>>(
            nthreads, bias_data, output_data, bias_input_size, bias_output_size);
    return 0;
}
