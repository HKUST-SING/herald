#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void conv2d_broadcast_to_kernel(size_t nthreads,
                                           const float *input_data,
                                           float *output_data,
                                           size_t input_size,
                                           size_t output_size) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= nthreads)
        return;
    size_t input_id = (id % (input_size * output_size)) / output_size;
    output_data[id] = input_data[input_id];
}

int DLGpuConv2d_broadcast_to(const DLArrayHandle input_x,
                             DLArrayHandle output_y,
                             DLStreamHandle stream_handle = NULL) {
    assert(input_x->shape[0] == output_y->shape[1]);
    const float *input_data = (const float *)input_x->data;
    float *output_data = (float *)output_y->data;
    size_t batch_size = output_y->shape[0];
    size_t input_size = input_x->shape[0];
    size_t output_size = (output_y->shape[2]) * (output_y->shape[3]);
    size_t nthreads = batch_size * input_size * output_size;
    size_t BLOCKS = (nthreads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    if (stream_handle)
        conv2d_broadcast_to_kernel<<<BLOCKS, THREADS_PER_BLOCK, 0,
                                     *(hipStream_t *)stream_handle->handle>>>(
            nthreads, input_data, output_data, input_size, output_size);
    else
        conv2d_broadcast_to_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(
            nthreads, input_data, output_data, input_size, output_size);
    return 0;
}